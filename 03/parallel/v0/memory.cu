#include "head.h"

double *h_t;
double *d_t;
double *h_V;
double *d_V;
//double *d_dV2;
double *h_Vnew;
//double *d_Vnew;
double *d_it;

double *h_m;
double *d_m;
double *h_h;
double *d_h;
double *h_jj;
double *d_jj;
double *h_d;
double *d_d;
double *h_f;
double *d_f;
double *h_X;
double *d_X;
double *h_cai;
double *d_cai;

double *h_it;

double *d_m0;
double *d_h0;
double *d_jj0;
double *d_d0;
double *d_f0;
double *d_X0;

double *d_dVdt;
double *dcai;

double *d_ADIf;
double *d_y;
double *d_belta;

void Allocate(){
	hipError_t Error;
	size_t size = nx*ny*sizeof(double);

	h_t = (double*)malloc(size);
	Error = hipMalloc((void**)&d_t, size);
	printf("CUDA error = %s\n",hipGetErrorString(Error));

	h_V = (double*)malloc(size);
	hipMalloc((void**)&d_V, size);
	//hipMalloc((void**)&d_dV2, size);
	h_Vnew = (double*)malloc(size);
	//hipMalloc((void**)&d_Vnew, size);

	hipMalloc((void**)&d_it, size);

	h_m = (double*)malloc(size);
	hipMalloc((void**)&d_m, size);
	h_h = (double*)malloc(size);
        hipMalloc((void**)&d_h, size);
	h_jj = (double*)malloc(size);
        hipMalloc((void**)&d_jj, size);
	h_d = (double*)malloc(size);
        hipMalloc((void**)&d_d, size);
	h_f = (double*)malloc(size);
        hipMalloc((void**)&d_f, size);
	h_X = (double*)malloc(size);
        hipMalloc((void**)&d_X, size);
	h_cai = (double*)malloc(size);
        hipMalloc((void**)&d_cai, size);

	h_it = (double*)malloc(size);

	hipMalloc((void**)&d_m0, size);
        hipMalloc((void**)&d_h0, size);
        hipMalloc((void**)&d_jj0, size);
        hipMalloc((void**)&d_d0, size);
        hipMalloc((void**)&d_f0, size);
        hipMalloc((void**)&d_X0, size);

	hipMalloc((void**)&d_dVdt, size);
	hipMalloc((void**)&dcai, size);

	hipMalloc((void**)&d_ADIf, size);
	hipMalloc((void**)&d_y, size);
	hipMalloc((void**)&d_belta, size);
}

void free(){

	free(h_t);free(h_V);free(h_m);free(h_h);
	free(h_jj);free(h_d);free(h_f);free(h_X);free(h_cai);
	free(h_Vnew);
	free(h_it);

	hipFree(d_t);hipFree(d_V);hipFree(d_it);//hipFree(d_dV2);hipFree(d_Vnew);
	hipFree(d_m);hipFree(d_h);hipFree(d_jj);hipFree(d_d);
	hipFree(d_f);hipFree(d_X);hipFree(d_cai);

	hipFree(d_m0);hipFree(d_h0);hipFree(d_jj0);hipFree(d_d0);
        hipFree(d_f0);hipFree(d_X0);hipFree(d_dVdt);hipFree(dcai);

	hipFree(d_ADIf);
}

void Send_to_Device(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

	Error = hipMemcpy(d_t, h_t, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_t->d_t) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(d_V, h_V, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_V->d_V) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_m, h_m, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_m->d_m) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_h, h_h, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_h->d_h) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_jj, h_jj, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_jj->d_jj) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_d->d_d) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_f->d_f) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_X->d_X) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_cai, h_cai, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_cai->d_cai) = %s\n",hipGetErrorString(Error));
}

void Send_to_Host(){
	hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_V, d_V, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_m, d_m, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_m->h_m) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_h, d_h, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_h->h_h) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_jj, d_jj, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_jj->h_jj) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_d, d_d, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_d->h_d) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_f, d_f, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_f->h_f) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_X, d_X, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_X->h_X) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_cai, d_cai, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_cai->h_cai) = %s\n",hipGetErrorString(Error));
}

void Send_V(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_V, d_V, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Vnew->Vnew) = %s\n",hipGetErrorString(Error));
/*
	Error = hipMemcpy(h_Vnew, d_ADIf, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Vnew->Vnew) = %s\n",hipGetErrorString(Error));
*/
}

void Save_Result(){

        FILE *pFile;
        int i,j;
        int index;
        //int n;
        //n = nx;
        pFile = fopen("Vnew.txt","w+");
        // Save the matrix V
        for (i = 0; i < ny; i++) {
                for (j = 0; j < nx; j++) {
                        index = i*nx + j;
                        fprintf(pFile, "%g", h_Vnew[index]);
                        if (j == (nx-1)) {
                                fprintf(pFile, "\n");
                        }else{
                                fprintf(pFile, "\t");
                        }
                }
        }
        fclose(pFile);

	pFile = fopen("V.txt","w+");
        // Save the matrix V
        for (i = 0; i < ny; i++) {
                for (j = 0; j < nx; j++) {
                        index = i*nx + j;
                        fprintf(pFile, "%g", h_V[index]);
                        if (j == (nx-1)) {
                                fprintf(pFile, "\n");
                        }else{
                                fprintf(pFile, "\t");
                        }
                }
        }
        fclose(pFile);
}

