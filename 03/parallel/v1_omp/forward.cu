#include "hip/hip_runtime.h"
#include "head.h"

#define tpb 256

extern double *d_t;
extern double *d_it;
extern double *d_V;
extern double *d_dV2;
extern double *d_Vnew;
extern double *d_m;
extern double *d_h;
extern double *d_jj;
extern double *d_d;
extern double *d_f;
extern double *d_X;
extern double *d_cai;

extern double *d_m0;
extern double *d_h0;
extern double *d_jj0;
extern double *d_d0;
extern double *d_f0;
extern double *d_X0;

extern double *d_dVdt;
extern double *dcai;

extern double *d_belta;
extern double *d_ADIf;
extern double *d_y;

/*
__global__ void boundary(double *d_V){
	int k = blockDim.x * blockIdx.x + threadIdx.x;

	if(k<nx){

	d_V[(k+1)*(nx+2)] = d_V[(k+1)*(nx+2)+1];
        d_V[(k+1)*(nx+2)+(nx+1)] = d_V[(k+1)*(nx+2)+nx];
        d_V[k+1] = d_V[k+1+(nx+2)];
        d_V[(ny+1)*(nx+2)+k+1] = d_V[ny*(nx+2)+k+1];

	}
}

void bc(){
        int bpg;
        //tpb = 256;
        bpg = (nx+tpb-1)/tpb;
        boundary<<<bpg, tpb>>>(d_V);
        //hipDeviceSynchronize();
}
*/
/*
__global__ void comp_dV2(double *d_V ,double *d_dV2){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){

	int i = (int)(k/nx);
	int id = k+(nx+2)+1+(2*i);

	d_dV2[k] = D*((d_V[id+1] + d_V[id-1] - 2*d_V[id]) / (dx*dx) + (d_V[id+(nx+2)] + d_V[id-(nx+2)] - 2*d_V[id])/(dy*dy));

	}
}

void dV2(){
	int bpg;
	//tpb = 256;
        bpg = (nx*ny+tpb-1)/tpb;
	comp_dV2<<<bpg, tpb>>>(d_V, d_dV2);
	//hipDeviceSynchronize();
}
*/
__device__ void comp_it(double *d_V, double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_cai, double *dcai, double *d_X, double *d_it, double *d_m0, double *d_h0, double *d_jj0, double *d_d0, double *d_f0, double *d_X0, int I, int i, int k, double *d_t) {
	//int id = k+nx+2+1+2*j;
	d_it[k] = 0.0;

	//comp_ina
	double gna = 23;
        double ena = ((R*temp) / frdy)*log(nao / nai);

        double am = 0.32*(d_V[k] + 47.13) / (1 - exp(-0.1*(d_V[k] + 47.13)));
        double bm = 0.08*exp(-d_V[k] / 11);
	double ah, bh, aj ,bj;
        if (d_V[k] < -40.0) {
                ah = 0.135*exp((80 + d_V[k]) / -6.8);
                bh = 3.56*exp(0.079*d_V[k]) + 310000 * exp(0.35*d_V[k]);
                aj = (-127140 * exp(0.2444*d_V[k]) - 0.00003474*exp(-0.04391*d_V[k]))*((d_V[k] + 37.78)/(1 + exp(0.311*(d_V[k] + 79.23))));
                bj = (0.1212*exp(-0.01052*d_V[k])) / (1 + exp(-0.1378*(d_V[k] + 40.14)));
        }
        else {
                ah = 0;
                bh = 1 / (0.13*(1 + exp((d_V[k] + 10.66) / -11.1)));
                aj = 0;
                bj = (0.3*exp(-0.0000002535*d_V[k])) / (1 + exp(-0.1*(d_V[k] + 32)));
        }
        double mtau = 1 / (am + bm);
        double htau = 1 / (ah + bh);
	double jtau = 1 / (aj + bj);

        double mss = am*mtau;
        double hss = ah*htau;
        double jss = aj*jtau;

        d_m0[k] = mss - (mss - d_m[k])*exp(-d_t[k] / mtau);
        d_h0[k] = hss - (hss - d_h[k])*exp(-d_t[k] / htau);
        d_jj0[k] = jss - (jss - d_jj[k])*exp(-d_t[k] / jtau);

        d_it[k] += gna*d_m0[k] * d_m0[k] * d_m0[k] * d_h0[k] * d_jj0[k] * (d_V[k] - ena);
	//comp_ical
	__shared__ double esi[tpb];
	__shared__ double isi[tpb];
        esi[I] = 7.7 - 13.0287*log(d_cai[k]);

        double ad = 50 * 0.095*exp(-0.01*(d_V[k] - 5)) / (1 + exp(-0.072*(d_V[k] - 5)));
        double bd = 50 * 0.07*exp(-0.017*(d_V[k] + 44)) / (1 + exp(0.05*(d_V[k] + 44)));
        double af = 50 * 0.012*exp(-0.008*(d_V[k] + 28)) / (1 + exp(0.15*(d_V[k] + 28)));
        double bf = 50 * 0.0065*exp(-0.02*(d_V[k] + 30)) / (1 + exp(-0.2*(d_V[k] + 30)));

        double taud = 1 / (ad + bd);
        double tauf = 1 / (af + bf);

        double dss = ad*taud;
        double fss = af*tauf;

        d_d0[k] = dss - (dss - d_d[k])*exp(-d_t[k] / taud);
        d_f0[k] = fss - (fss - d_f[k])*exp(-d_t[k] / tauf);

        isi[I] = 0.09*d_d0[k] * d_f0[k] * (d_V[k] - esi[I]);

        dcai[k] = -0.0001*isi[I] + 0.07*(0.0001 - d_cai[k]);

        //d_cai[k] = d_cai[k] + dcai*dt;
	d_it[k] = d_it[k] + isi[I];
	//comp_ik
        double gk = 0.282*sqrt(ko / 5.4);
        double ek = ((R*temp) / frdy)*log(ko / ki);
        //double prnak = 0.01833;
        //ek = ((R*temp) / frdy)*log((ko + prnak*nao) / (ki + prnak*nai));

        double ax = 50 * 0.0005*exp(0.083*(d_V[k] + 50)) / (1 + exp(0.057*(d_V[k] + 50)));
        double bx = 50 * 0.0013*exp(-0.06*(d_V[k] + 20)) / (1 + exp(-0.04*(d_V[k] + 20)));

        double taux = 1 / (ax + bx);
        double xss = ax*taux;
        d_X0[k] = xss - (xss - d_X[k])*exp(-d_t[k] / taux);

	double Xi;
        if (d_V[k] > -100) {
                Xi = 2.837*(exp(0.04*(d_V[k] + 77)) - 1)/((d_V[k] + 77)*exp(0.04*(d_V[k] + 35)));
        }
        else {
                Xi = 1;
        }
        d_it[k] += gk*d_X0[k] * Xi*(d_V[k] - ek);
	//comp_ik1
        double gk1 = 0.6047*(sqrt(ko / 5.4));
        double ek1 = ((R*temp) / frdy)*log(ko / ki);

        double ak1 = 1.02 / (1 + exp(0.2385*(d_V[k] - ek1 - 59.215)));
        double bk1 = (0.49124*exp(0.08032*(d_V[k] - ek1 + 5.476))+exp(0.06175*(d_V[k] - ek1 - 594.31)))/(1 + exp(-0.5143*(d_V[k] - ek1 + 4.753)));
        double K1ss = ak1 / (ak1 + bk1);

        d_it[k] += gk1*K1ss*(d_V[k] - ek1);
	//comp_ikp
        double gkp = 0.0183;
        double ekp = ((R*temp) / frdy)*log(ko / ki);

        double kp = 1 / (1 + exp((7.488 - d_V[k]) / 5.98));

        d_it[k] += gkp*kp*(d_V[k] - ekp);
	//comp_ib
        d_it[k] += 0.03921*(d_V[k] + 59.87);

}

__global__ void comp_dVdt(double *d_V, double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_cai, double *dcai, double *d_X, double *d_it, double *d_m0, double *d_h0, double *d_jj0, double *d_d0, double *d_f0, double *d_X0, double *d_dVdt, double *d_t){

	int k = threadIdx.x + blockIdx.x * blockDim.x;
        int I = threadIdx.x;

        if(k<nx*ny){

        int i = (int)(k/nx);
	d_t[k] = dt_max;
	comp_it(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, I, i, k, d_t);
	d_dVdt[k] = -d_it[k];
	}
}

void dVdt(){
	int bpg;

        bpg = (nx*ny+tpb-1)/tpb;
        comp_dVdt<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, d_dVdt, d_t);
}

__global__ void plane_waves(double *d_dVdt){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<ny*5){
	int i, j, id;
	i = (int)(k/5);
	j = k-i*5;
	id = i*nx+j;

	d_dVdt[id] = d_dVdt[id] + (-st);

	}
}

void stimu(){
	int bpg;
        //int tpb;

        //tpb = 256;
        bpg = (ny*5+tpb-1)/tpb;
	plane_waves<<<bpg, tpb>>>(d_dVdt);
	//hipDeviceSynchronize();
}


__device__ void gate(double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_X, double *d_m0, double *d_h0, double *d_jj0, double *d_d0, double *d_f0, double *d_X0, int k){
	d_m[k] = d_m0[k];
	d_h[k] = d_h0[k];
	d_jj[k] = d_jj0[k];
	d_d[k] = d_d0[k];
	d_f[k] = d_f0[k];
	d_X[k] = d_X0[k];
}

__global__ void comp_ODE_stim(double *d_V, double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_cai, double *dcai, double *d_X, double *d_it, double *d_m0, double *d_h0, double *d_jj0, double *d_d0, double *d_f0, double *d_X0, double *d_dVdt, double *d_t, int ncount){

	int k = threadIdx.x + blockIdx.x * blockDim.x;
        int I = threadIdx.x;

        if(k<nx*ny){

        int i = (int)(k/nx);
	int j = k - i*nx;
	int id = i*nx+j;
	int k1, k0, ttt;
	//int vid = (i+1)*(nx+2)+j+1;

	if(d_dVdt[id]>0){
		k0 = 5;
	}else{
		k0 = 1;
	}
	k1 = k0 + (int)(fabs(d_dVdt[id]) + 0.5);
	if (k1 >(int)(dt_max / dt_min)){
		k1 = (int)(dt_max / dt_min);
	}
	d_t[id] = dt_max / k1;
	for (ttt = 0; ttt < k1; ttt++){ //from t to t+dt_max, t=t+dt
		comp_it(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, I, i, id, d_t);
		gate(d_m, d_h, d_jj, d_d, d_f, d_X, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, id);
		d_cai[id] = d_cai[id] + dcai[id]*d_t[id];//renew Cai
		d_dVdt[id] = -d_it[id] + (-st)*(j>=0 && j<5 && ncount>=1 && ncount <= stimtime);
		d_V[id] = d_V[id] + d_t[id]*d_dVdt[id];
        }

	}
}

void ODE(int ncount){
	int bpg;
        bpg = (nx*ny+tpb-1)/tpb;
        comp_ODE_stim<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, d_dVdt, d_t, ncount);
	//bpg = ((nx-5)*ny+tpb-1)/tpb;
	//comp_ODE<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, d_dVdt, d_t, 5);
}

__global__ void sweep_x(double *d_V, double *d_ADIf){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){
	int i = (int)(k/nx);
	int j = k-nx*i;
	int id = i*nx+j;
	//d_ADIf[id] = 0.0;
	d_ADIf[id] = d_V[id]  + (dt_max*D / (dx*dx)/2)*((d_V[id] - 2 * d_V[id] + d_V[id+1])*(j==0)+(d_V[id-1] - 2 * d_V[id] + d_V[id])*(j==nx-1)+(d_V[id-1] - 2 * d_V[id] + d_V[id+1])*(j>0 && j<nx-1));
//	d_ADIf[k] = d_V[k]  + (dt_max*D / (dx*dx)/2)*((d_V[k] - 2 * d_V[k] + d_V[k+nx])*(k-nx*(int)(k/nx)==0)+(d_V[k-nx] - 2 * d_V[k] + d_V[k])*(k-nx*(int)(k/nx)==ny-1)+(d_V[k-nx] - 2 * d_V[k] + d_V[k+nx])*(k-nx*(int)(k/nx)>0 && k-nx*(int)(k/nx)<ny-1));
/*	if(j==0){
		d_ADIf[id] = d_V[id]  + (dt_max*D / (dx*dx)/2)*(d_V[id] - 2 * d_V[id] + d_V[id+1]);
	}else if(j==nx-1){
		d_ADIf[id] = d_V[id] + (dt_max*D / (dx*dx)/2)*(d_V[id-1] - 2 *d_V[id] + d_V[id]);
	}else{
		d_ADIf[id] = d_V[id] + (dt_max*D / (dx*dx)/2)*(d_V[id-1] - 2 * d_V[id] + d_V[id+1]);
	}*/

	}
/*
                for (j = 1; j < ny + 1; j++){
                        for (i = 1; i < nx + 1; i++){
                                if (j==1){
                                        f[i][j] = V[i][j]  + (eta/2)*(V[i][j] - 2 * V[i][j] + V[i][j + 1]);
                                }else if (j==ny){
                                        f[i][j] = V[i][j] + (eta/2)*(V[i][j - 1] - 2 * V[i][j] + V[i][j]);
                                }else{
                                        f[i][j] = V[i][j] + (eta/2)*(V[i][j - 1] - 2 * V[i][j] + V[i][j + 1]);
                                }
                        }
                }
*/
}

void x_direction(){

	int bpg;
        bpg = (nx*ny+tpb-1)/tpb;
	sweep_x<<<bpg, tpb>>>(d_V, d_ADIf);
}

__global__ void solve_x(double *d_V, double *d_ADIf, double *d_belta, double *d_y){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

        if(k<nx){
	double eta = dt_max*D / (dx*dx);
        double b = 1 + eta;
        double b_1 = 1 + eta / 2;//take care the boundary value
        double b_n = 1 + eta / 2;//take care the boundary value
        double c = -eta / 2;
        double a = -eta / 2;

	int i;

        d_belta[0+k] = c / b_1;//-dt_max*D / (2*dx*dx+dt_max*D);
        d_y[0+k] = d_ADIf[0+k] / b_1;//(1+dt_max*D / (2*dx*dx));
	for (i = 1; i < ny-1; i++){ //i = 2,3,...,n-1
        	d_belta[i*nx+k] = c/(b-a*d_belta[(i-1)*nx+k]);
        	d_y[i*nx+k] = (d_ADIf[i*nx+k] - a*d_y[(i-1)*nx+k]) / (b-a*d_belta[(i-1)*nx+k]);
        }
	d_y[(ny-1)*nx+k] = (d_ADIf[(ny-1)*nx+k] - a*d_y[(ny-1-1)*nx+k]) / (b_n - a*d_belta[(ny-1-1)*nx+k]);
        d_V[(ny-1)*nx+k] = d_y[(ny-1)*nx+k];
        for (i = ny-2; i >=0; i--){
        	d_V[i*nx+k] = d_y[i*nx+k] - d_belta[i*nx+k] * d_V[(i+1)*nx+k];
        }

	}

}

void x_solve(){

        int bpg;
        bpg = (nx+tpb-1)/tpb;
        solve_x<<<bpg, tpb>>>(d_V, d_ADIf, d_belta, d_y);
}

__global__ void sweep_y(double *d_V, double *d_ADIf){
        int k = threadIdx.x + blockIdx.x * blockDim.x;

        if(k<nx*ny){
        int i = (int)(k/nx);
        int j = k-nx*i;
        int id = i*nx+j;
        //d_ADIf[id] = 0.0;
        d_ADIf[id] = d_V[id]  + (dt_max*D / (dx*dx)/2)*((d_V[id] - 2 * d_V[id] + d_V[id+nx])*(i==0)+(d_V[id-nx] - 2 * d_V[id] + d_V[id])*(i==ny-1)+(d_V[id-nx] - 2 * d_V[id] + d_V[id+nx])*(i>0 && i<ny-1));
	}
}

void y_direction(){

        int bpg;
        bpg = (nx*ny+tpb-1)/tpb;
        sweep_y<<<bpg, tpb>>>(d_V, d_ADIf);
}

__global__ void solve_y(double *d_V, double *d_ADIf, double *d_belta, double *d_y){
        int k = threadIdx.x + blockIdx.x * blockDim.x;

        if(k<ny){
        double eta = dt_max*D / (dx*dx);
        double b = 1 + eta;
        double b_1 = 1 + eta / 2;//take care the boundary value
        double b_n = 1 + eta / 2;//take care the boundary value
        double c = -eta / 2;
        double a = -eta / 2;

        int j;

        d_belta[0+nx*k] = c / b_1;//-dt_max*D / (2*dx*dx+dt_max*D);
        d_y[0+nx*k] = d_ADIf[0+nx*k] / b_1;//(1+dt_max*D / (2*dx*dx));
        for (j = 1; j < nx-1; j++){ //i = 2,3,...,n-1
                d_belta[k*nx+j] = c/(b-a*d_belta[k*nx+j-1]);
                d_y[k*nx+j] = (d_ADIf[k*nx+j] - a*d_y[k*nx+j-1]) / (b-a*d_belta[k*nx+j-1]);
        }
        d_y[k*nx+nx-1] = (d_ADIf[k*nx+nx-1] - a*d_y[k*nx+nx-1-1]) / (b_n - a*d_belta[k*nx+nx-1-1]);
        d_V[k*nx+nx-1] = d_y[k*nx+nx-1];
        for (j = ny-2; j >=0; j--){
                d_V[k*nx+j] = d_y[k*nx+j] - d_belta[k*nx+j] * d_V[k*nx+j+1];
        }

        }

}

void y_solve(){

        int bpg;
        bpg = (ny+tpb-1)/tpb;
        solve_y<<<bpg, tpb>>>(d_V, d_ADIf, d_belta, d_y);
}
/*
__global__ void Euler(double *d_V, double *d_dV2, double *d_Vnew){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){

	int i = (int)(k/nx);
	d_Vnew[k] = d_V[k+nx+2+1+2*i] + dt_max/2 *d_dV2[k];
        d_V[k+nx+2+1+2*i] = d_Vnew[k];

	}
}

void Forward_Euler(){
	int bpg;
        //int tpb;

        //tpb = 256;
        bpg = (nx*ny+tpb-1)/tpb;
	Euler<<<bpg, tpb>>>(d_V, d_dV2, d_Vnew);
	//hipDeviceSynchronize();
}
*/
