#include "hip/hip_runtime.h"
#include "head.h"

#define tpb 256

extern double *d_t;
extern double *d_it;
extern double *d_V;
extern double *d_dV2;
extern double *d_Vnew;
extern double *d_m;
extern double *d_h;
extern double *d_jj;
extern double *d_d;
extern double *d_f;
extern double *d_X;
extern double *d_cai;
/*
extern double *d_m0;
extern double *d_h0;
extern double *d_jj0;
extern double *d_d0;
extern double *d_f0;
extern double *d_X0;
*/
extern double *d_dVdt;
//extern double *dcai;
extern double *d_isi;
extern double *d_D1V;
extern double *d_D2V;

//extern int ncount;
__global__ void boundary(double *d_V){
	int k = blockDim.x * blockIdx.x + threadIdx.x;

	if(k<nx){

	d_V[(k+1)*(nx+2)] = d_V[(k+1)*(nx+2)+1];
        d_V[(k+1)*(nx+2)+(nx+1)] = d_V[(k+1)*(nx+2)+nx];
        d_V[k+1] = d_V[k+1+(nx+2)];
        d_V[(ny+1)*(nx+2)+k+1] = d_V[ny*(nx+2)+k+1];

	}
}

void bc(){
        int bpg;
        //tpb = 256;
        bpg = (nx+tpb-1)/tpb;
        boundary<<<bpg, tpb>>>(d_V);
        //hipDeviceSynchronize();
}


__global__ void comp_dV2(double *d_V ,double *d_dV2){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){

	int i = (int)(k/nx);
	int id = k+(nx+2)+1+(2*i);

	d_dV2[k] = D*((d_V[id+1] + d_V[id-1] - 2*d_V[id]) / (dx*dx) + (d_V[id+(nx+2)] + d_V[id-(nx+2)] - 2*d_V[id])/(dy*dy));

	}
}

void dV2(){
	int bpg;
	//tpb = 256;
        bpg = (nx*ny+tpb-1)/tpb;
	comp_dV2<<<bpg, tpb>>>(d_V, d_dV2);
	//hipDeviceSynchronize();
}

__device__ void comp_it(double *d_V, double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_cai, double *d_isi, double *d_X, double *d_it, int I, int i, int k, double *d_t) {
	d_it[k] = 0.0;

	int id = k+nx+2+1+2*i;

	//comp_ina
	//double gna = 23;
        //double ena = ((R*temp) / frdy)*log(nao / nai);
	/*
        double am = 0.32*(d_V[k+nx+2+1+2*i] + 47.13) / (1 - exp(-0.1*(d_V[k+nx+2+1+2*i] + 47.13)));
        double bm = 0.08*exp(-d_V[k+nx+2+1+2*i] / 11);
	double ah, bh, aj ,bj;
        if (d_V[k+nx+2+1+2*i] < -40.0) {
                ah = 0.135*exp((80 + d_V[k+nx+2+1+2*i]) / -6.8);
                bh = 3.56*exp(0.079*d_V[k+nx+2+1+2*i]) + 310000 * exp(0.35*d_V[k+nx+2+1+2*i]);
                aj = (-127140 * exp(0.2444*d_V[k+nx+2+1+2*i]) - 0.00003474*exp(-0.04391*d_V[k+nx+2+1+2*i]))*
                        ((d_V[k+nx+2+1+2*i] + 37.78)/(1 + exp(0.311*(d_V[k+nx+2+1+2*i] + 79.23))));
                bj = (0.1212*exp(-0.01052*d_V[k+nx+2+1+2*i])) / (1 + exp(-0.1378*(d_V[k+nx+2+1+2*i] + 40.14)));
        }
        else {
                ah = 0;
                bh = 1 / (0.13*(1 + exp((d_V[k+nx+2+1+2*i] + 10.66) / -11.1)));
                aj = 0;
                bj = (0.3*exp(-0.0000002535*d_V[k+nx+2+1+2*i])) / (1 + exp(-0.1*(d_V[k+nx+2+1+2*i] + 32)));
        }
        double mtau = 1 / (am + bm);
        double htau = 1 / (ah + bh);
	double jtau = 1 / (aj + bj);

        double mss = am*mtau;
        double hss = ah*htau;
        double jss = aj*jtau;

        d_m0[k] = mss - (mss - d_m[k])*exp(-d_t[k] / mtau);
        d_h0[k] = hss - (hss - d_h[k])*exp(-d_t[k] / htau);
        d_jj0[k] = jss - (jss - d_jj[k])*exp(-d_t[k] / jtau);
	*/
        d_it[k] += gna*d_m[k] * d_m[k] * d_m[k] * d_h[k] * d_jj[k] * (d_V[id] - ena);
	//comp_ical
	__shared__ double esi[tpb];
	//__shared__ double isi[tpb];
        esi[I] = 7.7 - 13.0287*log(d_cai[k]);
	/*
        double ad = 50 * 0.095*exp(-0.01*(d_V[k+nx+2+1+2*i] - 5)) / (1 + exp(-0.072*(d_V[k+nx+2+1+2*i] - 5)));
        double bd = 50 * 0.07*exp(-0.017*(d_V[k+nx+2+1+2*i] + 44)) / (1 + exp(0.05*(d_V[k+nx+2+1+2*i] + 44)));
        double af = 50 * 0.012*exp(-0.008*(d_V[k+nx+2+1+2*i] + 28)) / (1 + exp(0.15*(d_V[k+nx+2+1+2*i] + 28)));
        double bf = 50 * 0.0065*exp(-0.02*(d_V[k+nx+2+1+2*i] + 30)) / (1 + exp(-0.2*(d_V[k+nx+2+1+2*i] + 30)));

        double taud = 1 / (ad + bd);
        double tauf = 1 / (af + bf);

        double dss = ad*taud;
        double fss = af*tauf;

        d_d0[k] = dss - (dss - d_d[k])*exp(-d_t[k] / taud);
        d_f0[k] = fss - (fss - d_f[k])*exp(-d_t[k] / tauf);
	*/
        d_isi[k] = 0.09*d_d[k] * d_f[k] * (d_V[id] - esi[I]);

        //dcai[k] = -0.0001*isi[I] + 0.07*(0.0001 - d_cai[k]);

        //d_cai[k] = d_cai[k] + dcai*dt;
	d_it[k] = d_it[k] + d_isi[k];
	//comp_ik
	/*
        double gk = 0.282*sqrt(ko / 5.4);
        double ek = ((R*temp) / frdy)*log(ko / ki);
        //double prnak = 0.01833;
        //ek = ((R*temp) / frdy)*log((ko + prnak*nao) / (ki + prnak*nai));

        double ax = 50 * 0.0005*exp(0.083*(d_V[k+nx+2+1+2*i] + 50)) / (1 + exp(0.057*(d_V[k+nx+2+1+2*i] + 50)));
        double bx = 50 * 0.0013*exp(-0.06*(d_V[k+nx+2+1+2*i] + 20)) / (1 + exp(-0.04*(d_V[k+nx+2+1+2*i] + 20)));

        double taux = 1 / (ax + bx);
        double xss = ax*taux;
        d_X0[k] = xss - (xss - d_X[k])*exp(-d_t[k] / taux);
	*/
	double Xi;
        if (d_V[id] > -100) {
                Xi = 2.837*(exp(0.04*(d_V[id] + 77)) - 1)/((d_V[id] + 77)*exp(0.04*(d_V[id] + 35)));
        }
        else {
                Xi = 1;
        }
        d_it[k] += gk*d_X[k] * Xi*(d_V[id] - ek);
	//comp_ik1
        //double gk1 = 0.6047*(sqrt(ko / 5.4));
        //double ek1 = ((R*temp) / frdy)*log(ko / ki);

        double ak1 = 1.02 / (1 + exp(0.2385*(d_V[id] - ek1 - 59.215)));
        double bk1 = (0.49124*exp(0.08032*(d_V[id] - ek1 + 5.476))+exp(0.06175*(d_V[id] - ek1 - 594.31)))/(1 + exp(-0.5143*(d_V[id] - ek1 + 4.753)));
        double K1ss = ak1 / (ak1 + bk1);

        d_it[k] += gk1*K1ss*(d_V[id] - ek1);
	//comp_ikp
        //double gkp = 0.0183;
        //double ekp = ((R*temp) / frdy)*log(ko / ki);

        double kp = 1 / (1 + exp((7.488 - d_V[id]) / 5.98));

        d_it[k] += gkp*kp*(d_V[id] - ekp);
	//comp_ib
        d_it[k] += 0.03921*(d_V[id] + 59.87);

}

__global__ void comp_dVdt(double *d_V, double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_cai, double *d_isi, double *d_X, double *d_it, double *d_dVdt, double *d_t){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
        int I = threadIdx.x;

        if(k<nx*ny){

        int i = (int)(k/nx);
	d_t[k] = dt_max;
	comp_it(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, d_isi, d_X, d_it, I, i, k, d_t);
	d_dVdt[k] = -d_it[k];
	}
}

void dVdt(){
	int bpg;

        bpg = (nx*ny+tpb-1)/tpb;
        comp_dVdt<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, d_isi, d_X, d_it, d_dVdt, d_t);
}

__global__ void plane_waves(double *d_dVdt){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<ny*5){
	int i, j, id;
	i = (int)(k/5);
	j = k-i*5;
	id = i*nx+j;

	d_dVdt[id] = d_dVdt[id] + (-st);

	}
}

void stimu(){
	int bpg;
        //int tpb;

        //tpb = 256;
        bpg = (ny*5+tpb-1)/tpb;
	plane_waves<<<bpg, tpb>>>(d_dVdt);
	//hipDeviceSynchronize();
}

/* Calculating the initial Ions Current*/
__device__ void comp_fluxes(double *d_cai, double *d_isi, int k, double *d_t){
        d_cai[k] = d_cai[k] + (-0.0001*d_isi[k] + 0.07*(0.0001 - d_cai[k]))*d_t[k];
}

/* update the gate value*/
__device__ void Rush_Larsen(double *d_V, double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_X, int i, int k, double *d_t){
	//Fast sodium current
        //gate variables can not be shared, should be local due to data racing !!!!!!!!
	int id = k+nx+2+1+2*i;
	//double bm = 0.08*exp(-d_V[k+nx+2+1+2*i] / 11);
	double am = 0.32*(d_V[id] + 47.13) / (1 - exp(-0.1*(d_V[id] + 47.13)));
        double bm = 0.08*exp(-d_V[id] / 11);
        double ah, bh, aj, bj;
        if (d_V[id] < -40.0) {
                ah = 0.135*exp((80 + d_V[id]) / -6.8);
                bh = 3.56*exp(0.079*d_V[id]) + 310000.0 * exp(0.35*d_V[id]);
                aj = (-127140 * exp(0.2444*d_V[id]) - 0.00003474*exp(-0.04391*d_V[id]))*((d_V[id] + 37.78) / (1.0 + exp(0.311*(d_V[id] + 79.23))));
                bj = (0.1212*exp(-0.01052*d_V[id])) / (1.0 + exp(-0.1378*(d_V[id] + 40.14)));
        }
        else {
                ah = 0.0;
                bh = 1.0 / (0.13*(1.0 + exp((d_V[id] + 10.66) / -11.1)));
		aj = 0.0;
                bj = (0.3*exp(-0.0000002535*d_V[id])) / (1.0 + exp(-0.1*(d_V[id] + 32.0)));
        }
        double mtau = 1.0 / (am + bm);
        double htau = 1.0 / (ah + bh);
        double jtau = 1.0 / (aj + bj);

        double mss = am*mtau;
        double hss = ah*htau;
        double jss = aj*jtau;
        d_m[k] = mss - (mss - d_m[k])*exp(-d_t[k] / mtau);
	d_h[k] = hss - (hss - d_h[k])*exp(-d_t[k] / htau);
        d_jj[k] = jss - (jss - d_jj[k])*exp(-d_t[k] / jtau);


        //Slow inward current
        double ad = 50*0.095*exp(-0.01*(d_V[id] - 5)) / (1.0 + exp(-0.072*(d_V[id] - 5)));
        double bd = 50*0.07*exp(-0.017*(d_V[id] + 44)) / (1.0 + exp(0.05*(d_V[id] + 44)));
        double af = 50*0.012*exp(-0.008*(d_V[id] + 28)) / (1.0 + exp(0.15*(d_V[id] + 28)));
        double bf = 50*0.0065*exp(-0.02*(d_V[id] + 30)) / (1.0 + exp(-0.2*(d_V[id] + 30)));

        double taud = 1.0 / (ad + bd);
	double tauf = 1.0 / (af + bf);

        double dss = ad*taud;
        double fss = af*tauf;
        d_d[k] = dss - (dss - d_d[k])*exp(-d_t[k] / taud);
        d_f[k] = fss - (fss - d_f[k])*exp(-d_t[k] / tauf);

        //Time-dependent potassium current
        double ax = 50*0.0005*exp(0.083*(d_V[id] + 50)) / (1 + exp(0.057*(d_V[id] + 50)));
        double bx = 50*0.0013*exp(-0.06*(d_V[id] + 20)) / (1 + exp(-0.04*(d_V[id] + 20)));
        double taux = 1 / (ax + bx);
	double xss = ax*taux;
        d_X[k] = xss - (xss - d_X[k])*exp(-d_t[k] / taux);
}

__global__ void firsttime(double *d_V, double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_X, double *d_cai, double *d_isi, double *d_dVdt, double *d_D1V, double *d_t){
	// in order to get D1V[i][j], for computing D2V[i][j] in CCL(i, j, dt_max);
	/* The first time step*/
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int i = (int)(k/nx);
	int id = k+nx+2+1+2*i;

	if(k<nx*ny){
		d_D1V[k] = d_dVdt[k];
		comp_fluxes(d_cai, d_isi, k, d_t);
        	Rush_Larsen(d_V, d_m, d_h, d_jj, d_d, d_f, d_X, i, k, d_t);
        	d_V[id] = d_V[id] + dt_max * d_dVdt[k];
	}
}

void First(){
        int bpg;

        bpg = (nx*ny+tpb-1)/tpb;
        firsttime<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_X, d_cai, d_isi, d_dVdt, d_D1V, d_t);
        //hipDeviceSynchronize();
}

__device__ void CCL(double *d_dVdt, double *d_D1V, double *d_D2V, int k, double *d_t){
/*
	double dt_range;
	dt_range = d_t[k]*2*(dt_univ > d_t[k]*2) + dt_univ*(dt_univ <= d_t[k]*2);

	d_D2V[k] = (d_dVdt[k] - d_D1V[k]) / d_t[k];
        double DiscriminantP = 0.0, DiscriminantN = 0.0, dtz = 0.0;
	DiscriminantP = d_dVdt[k] * d_dVdt[k] + 2 * d_D2V[k] * Voffset*(d_dVdt[k] >= 0);
        DiscriminantN = d_dVdt[k] * d_dVdt[k] - 2 * d_D2V[k] * Voffset*(d_dVdt[k] < 0);
        dtz = -d_dVdt[k] / d_D2V[k];
        d_t[k] = (-d_dVdt[k] + sqrt(DiscriminantP)) / d_D2V[k]*(d_dVdt[k] >= 0)*(d_D2V[k]>0)+(-d_dVdt[k] + sqrt(DiscriminantP)) / d_D2V[k]*(d_dVdt[k] >= 0)*(d_D2V[k]<0)*(DiscriminantP >= 0)+dtz*(d_dVdt[k] >= 0)*(d_D2V[k]<0)*(DiscriminantP < 0)+(-d_dVdt[k]+sqrt(DiscriminantN)) / d_D2V[k]*(d_dVdt[k] < 0)*(d_D2V[k]>0)*(DiscriminantN >= 0)+dtz*(d_dVdt[k] < 0)*(d_D2V[k]>0)*(DiscriminantN < 0)+(-d_dVdt[k] + sqrt(DiscriminantN)) / d_D2V[k]*(d_dVdt[k] < 0)*(d_D2V[k]<0);

        d_t[k] = d_t[k]*(d_t[k]<=dt_range && d_t[k]>=dt_min)+dt_range*(d_t[k]>dt_range)+dt_min*(d_t[k]<dt_min);
*/
	double dt_range;
        if (dt_univ > d_t[k] * 2){
                dt_range = d_t[k] * 2;
        }
        else{
                dt_range = dt_univ;
        }

        d_D2V[k] = (d_dVdt[k] - d_D1V[k]) / d_t[k];
        double DiscriminantP = 0, DiscriminantN = 0, dtz = 0;
        if (d_dVdt[k] >= 0){
                DiscriminantP = d_dVdt[k] * d_dVdt[k] + 2 * d_D2V[k] * Voffset;
                if (d_D2V[k]>0){
                        d_t[k] = (-d_dVdt[k] + sqrt(DiscriminantP)) / d_D2V[k];
                }
                else if (d_D2V[k]<0){
                        dtz = -d_dVdt[k] / d_D2V[k];
                        if (DiscriminantP >= 0){
                                d_t[k] = (-d_dVdt[k] + sqrt(DiscriminantP)) / d_D2V[k];
                        }
                        else{
                                d_t[k] = dtz;
                        }
                }
        }
	else{
                DiscriminantN = d_dVdt[k] * d_dVdt[k] - 2 * d_D2V[k] * Voffset;
                if (d_D2V[k]>0){
                        dtz = -d_dVdt[k] / d_D2V[k];
                        if (DiscriminantN >= 0){
                                d_t[k] = (-d_dVdt[k] - sqrt(DiscriminantN)) / d_D2V[k];
                        }
                        else{
                                d_t[k] = dtz;
                        }
                }
	else if (d_D2V[k]<0){
                        d_t[k] = (-d_dVdt[k] - sqrt(DiscriminantN)) / d_D2V[k];
                }
        }

        if (d_t[k]>dt_range){
                d_t[k] = dt_range;
        }
        if (d_t[k]<dt_min){
                d_t[k] = dt_min;
        }

}

__device__ void CCL_dtmax(double *d_dVdt, double *d_D1V, double *d_D2V, int k, double *d_t){
	double dt_range;
        if (dt_univ > dt_max * 2){
                dt_range = dt_max * 2;
        }
        else{
                dt_range = dt_univ;
        }

        d_D2V[k] = (d_dVdt[k] - d_D1V[k]) / dt_max;
        double DiscriminantP = 0, DiscriminantN = 0, dtz = 0;
        if (d_dVdt[k] >= 0){
                DiscriminantP = d_dVdt[k] * d_dVdt[k] + 2 * d_D2V[k] * Voffset;
                if (d_D2V[k]>0){
                        d_t[k] = (-d_dVdt[k] + sqrt(DiscriminantP)) / d_D2V[k];
                }
                else if (d_D2V[k]<0){
                        dtz = -d_dVdt[k] / d_D2V[k];
                        if (DiscriminantP >= 0){
                                d_t[k] = (-d_dVdt[k] + sqrt(DiscriminantP)) / d_D2V[k];
                        }
                        else{
                                d_t[k] = dtz;
                        }
                }
        }
	else{
                DiscriminantN = d_dVdt[k] * d_dVdt[k] - 2 * d_D2V[k] * Voffset;
                if (d_D2V[k]>0){
                        dtz = -d_dVdt[k] / d_D2V[k];
                        if (DiscriminantN >= 0){
                                d_t[k] = (-d_dVdt[k] - sqrt(DiscriminantN)) / d_D2V[k];
                        }
                        else{
                                d_t[k] = dtz;
                        }
                }
	else if (d_D2V[k]<0){
                        d_t[k] = (-d_dVdt[k] - sqrt(DiscriminantN)) / d_D2V[k];
                }
        }

        if (d_t[k]>dt_range){
                d_t[k] = dt_range;
        }
        if (d_t[k]<dt_min){
                d_t[k] = dt_min;
        }

}

__global__ void ODE_CCL(double *d_V, double *d_m, double *d_h, double *d_jj, double *d_d, double *d_f, double *d_cai, double *d_isi, double *d_X, double *d_it, double *d_dVdt, double *d_D1V, double *d_D2V, double *d_t, int ncount){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int i = (int)(k/nx);
	int j = k - i*nx;
	int id = k+nx+2+1+2*i;
	int I = threadIdx.x;

	double dt_sum;
	if(k<nx*ny){
		//***** adjust or correct time step---CCL method **/
                dt_sum = 0.0;
                CCL_dtmax(d_dVdt, d_D1V, d_D2V, k, d_t);
                dt_sum = dt_sum + d_t[k];
                while(dt_sum<dt_max){
                	d_D1V[k] = d_dVdt[k];
                	comp_fluxes(d_cai, d_isi, k, d_t);
                	Rush_Larsen(d_V, d_m, d_h, d_jj, d_d, d_f, d_X, i, k, d_t);
                	d_V[id] = d_V[id] + d_t[k] * d_dVdt[k] + d_t[k] * d_t[k] * d_D2V[k] / 2;
                	comp_it(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, d_isi, d_X, d_it, I, i, k, d_t);
                	d_dVdt[k] = -d_it[k] + (-st)*(ncount >= 1 && ncount <= stimtime && j >= 0 && j <= 4);
                	CCL(d_dVdt, d_D1V, d_D2V, k, d_t);
			dt_sum = dt_sum + d_t[k];
                }
                d_t[k] = dt_max - (dt_sum - d_t[k]);// here is a new dt  !!!
                d_D1V[k] = d_dVdt[k];
                comp_fluxes(d_cai, d_isi, k, d_t);
                Rush_Larsen(d_V, d_m, d_h, d_jj, d_d, d_f, d_X, i, k, d_t);
                d_V[id] = d_V[id] + d_t[k] * d_dVdt[k] + d_t[k] * d_t[k] * d_D2V[k] / 2;
	}
}

void ODE(int ncount){
        int bpg;

        bpg = (nx*ny+tpb-1)/tpb;
        ODE_CCL<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, d_isi, d_X, d_it, d_dVdt, d_D1V, d_D2V, d_t, ncount);
        //hipDeviceSynchronize();
}

__global__ void Euler(double *d_V, double *d_dV2, double *d_Vnew){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){

	int i = (int)(k/nx);
	d_Vnew[k] = d_V[k+nx+2+1+2*i] + dt_max/2 *d_dV2[k];
        d_V[k+nx+2+1+2*i] = d_Vnew[k];

	}
}

void Forward_Euler(){
	int bpg;
        //int tpb;

        //tpb = 256;
        bpg = (nx*ny+tpb-1)/tpb;
	Euler<<<bpg, tpb>>>(d_V, d_dV2, d_Vnew);
	//hipDeviceSynchronize();
}
