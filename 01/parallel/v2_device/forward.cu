#include "hip/hip_runtime.h"
#include "head.h"

#define tpb 256

extern double *d_t;
extern double *d_it;
extern double *d_V;
extern double *d_dV2;
extern double *d_Vnew;
extern double *d_m;
extern double *d_h;
extern double *d_jj;
extern double *d_d;
extern double *d_f;
extern double *d_X;
extern double *d_cai;

__global__ void boundary(double *d_V){
	int k = blockDim.x * blockIdx.x + threadIdx.x;

	if(k<nx){

	d_V[(k+1)*(nx+2)] = d_V[(k+1)*(nx+2)+1];
        d_V[(k+1)*(nx+2)+(nx+1)] = d_V[(k+1)*(nx+2)+nx];
        d_V[k+1] = d_V[k+1+(nx+2)];
        d_V[(ny+1)*(nx+2)+k+1] = d_V[ny*(nx+2)+k+1];

	}
}

__device__ void comp_it(double *d_V, double *d_m, double *d_h, double *d_jj,
                                 double *d_d, double *d_f, double *d_cai,
                                         double *d_X, double *d_it, int k, int j, int i) {

	//int id = k+nx+2+1+2*j;

	d_it[k] = 0.0;

	double gna = 23.0;
        double ena = ((R*temp) / frdy)*log(nao / nai);

        double am = 0.32*(d_V[k+nx+2+1+2*j] + 47.13) / (1 - exp(-0.1*(d_V[k+nx+2+1+2*j] + 47.13)));
        double bm = 0.08*exp(-d_V[k+nx+2+1+2*j] / 11);
	double ah, bh, aj ,bj;
        if (d_V[k+nx+2+1+2*j] < -40.0) {
                ah = 0.135*exp((80 + d_V[k+nx+2+1+2*j]) / -6.8);
                bh = 3.56*exp(0.079*d_V[k+nx+2+1+2*j]) + 310000 * exp(0.35*d_V[k+nx+2+1+2*j]);
                aj = (-127140 * exp(0.2444*d_V[k+nx+2+1+2*j]) - 0.00003474*exp(-0.04391*d_V[k+nx+2+1+2*j]))*
                        ((d_V[k+nx+2+1+2*j] + 37.78)/(1 + exp(0.311*(d_V[k+nx+2+1+2*j] + 79.23))));
                bj = (0.1212*exp(-0.01052*d_V[k+nx+2+1+2*j])) / (1 + exp(-0.1378*(d_V[k+nx+2+1+2*j] + 40.14)));
        }
        else {
                ah = 0;
                bh = 1 / (0.13*(1 + exp((d_V[k+nx+2+1+2*j] + 10.66) / -11.1)));
                aj = 0;
                bj = (0.3*exp(-0.0000002535*d_V[k+nx+2+1+2*j])) / (1 + exp(-0.1*(d_V[k+nx+2+1+2*j] + 32)));
        }
        double mtau = 1 / (am + bm);
        double htau = 1 / (ah + bh);
	double jtau = 1 / (aj + bj);

        double mss = am*mtau;
        double hss = ah*htau;
        double jss = aj*jtau;

        d_m[k] = mss - (mss - d_m[k])*exp(-dt / mtau);
        d_h[k] = hss - (hss - d_h[k])*exp(-dt / htau);
        d_jj[k] = jss - (jss - d_jj[k])*exp(-dt / jtau);

        //d_it[k] += gna*d_m[k] * d_m[k] * d_m[k] * d_h[k] * d_jj[k] * (d_V[k+nx+2+1+2*j] - ena);

	__shared__ double esi[tpb];
	__shared__ double isi[tpb];
        esi[i] = 7.7 - 13.0287*log(d_cai[k]);

        double ad = 0.095*exp(-0.01*(d_V[k+nx+2+1+2*j] - 5)) / (1 + exp(-0.072*(d_V[k+nx+2+1+2*j] - 5)));
        double bd = 0.07*exp(-0.017*(d_V[k+nx+2+1+2*j] + 44)) / (1 + exp(0.05*(d_V[k+nx+2+1+2*j] + 44)));
        double af = 0.012*exp(-0.008*(d_V[k+nx+2+1+2*j] + 28)) / (1 + exp(0.15*(d_V[k+nx+2+1+2*j] + 28)));
        double bf = 0.0065*exp(-0.02*(d_V[k+nx+2+1+2*j] + 30)) / (1 + exp(-0.2*(d_V[k+nx+2+1+2*j] + 30)));

        double taud = 1 / (ad + bd);
        double tauf = 1 / (af + bf);

        double dss = ad*taud;
        double fss = af*tauf;

        d_d[k] = dss - (dss - d_d[k])*exp(-dt / taud);
        d_f[k] = fss - (fss - d_f[k])*exp(-dt / tauf);

        isi[i] = 0.09*d_d[k] * d_f[k] * (d_V[k+nx+2+1+2*j] - esi[i]);

        double dcai = -0.0001*isi[i] + 0.07*(0.0001 - d_cai[k]);

        d_cai[k] = d_cai[k] + dcai*dt;
	//d_it[k] = d_it[k] + isi[i];

        double gk = 0.282*sqrt(ko / 5.4);
        double ek = ((R*temp) / frdy)*log(ko / ki);
        //double prnak = 0.01833;
        //ek = ((R*temp) / frdy)*log((ko + prnak*nao) / (ki + prnak*nai));

        double ax = 0.0005*exp(0.083*(d_V[k+nx+2+1+2*j] + 50)) / (1 + exp(0.057*(d_V[k+nx+2+1+2*j] + 50)));
        double bx = 0.0013*exp(-0.06*(d_V[k+nx+2+1+2*j] + 20)) / (1 + exp(-0.04*(d_V[k+nx+2+1+2*j] + 20)));

        double taux = 1 / (ax + bx);
        double xss = ax*taux;
        d_X[k] = xss - (xss - d_X[k])*exp(-dt / taux);

	double Xi;
        if (d_V[k+nx+2+1+2*j] > -100) {
                Xi = 2.837*(exp(0.04*(d_V[k+nx+2+1+2*j] + 77)) - 1)/
			((d_V[k+nx+2+1+2*j] + 77)*exp(0.04*(d_V[k+nx+2+1+2*j] + 35)));
        }
        else {
                Xi = 1;
        }
        //d_it[k] += gk*d_X[k] * Xi*(d_V[k+nx+2+1+2*j] - ek);

        double gk1 = 0.6047*(sqrt(ko / 5.4));
        double ek1 = ((R*temp) / frdy)*log(ko / ki);

        double ak1 = 1.02 / (1 + exp(0.2385*(d_V[k+nx+2+1+2*j] - ek1 - 59.215)));
        double bk1 = (0.49124*exp(0.08032*(d_V[k+nx+2+1+2*j] - ek1 + 5.476))+
			exp(0.06175*(d_V[k+nx+2+1+2*j] - ek1 - 594.31)))
                	/(1 + exp(-0.5143*(d_V[k+nx+2+1+2*j] - ek1 + 4.753)));
        double K1ss = ak1 / (ak1 + bk1);

        //d_it[k] += gk1*K1ss*(d_V[k+nx+2+1+2*j] - ek1);

        double gkp = 0.0183;
        double ekp = ((R*temp) / frdy)*log(ko / ki);

        double kp = 1 / (1 + exp((7.488 - d_V[k+nx+2+1+2*j]) / 5.98));

        //d_it[k] += gkp*kp*(d_V[k+nx+2+1+2*j] - ekp);

        //d_it[k] += 0.03921*(d_V[k+nx+2+1+2*j] + 59.87);
	d_it[k] = d_it[k] + gna*d_m[k] * d_m[k] * d_m[k] * d_h[k] * d_jj[k] * (d_V[k+nx+2+1+2*j] - ena)
			+ isi[i]
			+ gk*d_X[k] * Xi*(d_V[k+nx+2+1+2*j] - ek)
			+ gk1*K1ss*(d_V[k+nx+2+1+2*j] - ek1)
			+ gkp*kp*(d_V[k+nx+2+1+2*j] - ekp)
			+ 0.03921*(d_V[k+nx+2+1+2*j] + 59.87);
}

__global__ void comp(double *d_V, double *d_m, double *d_h, double *d_jj,
			 double *d_it, double *d_d, double *d_f, double *d_cai,
				double *d_X, double *d_dV2){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.x;

        if(k<nx*ny){

        int j = (int)(k/nx);

	comp_it(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, d_X, d_it, k, j, i);
	}
}

__global__ void comp_dV2(double *d_V ,double *d_dV2  ,double *d_it){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){

	int j = (int)(k/nx);

	int id = k+(nx+2)+1+(2*j);

	d_dV2[k] = -d_it[k] + D*((d_V[id+1] + d_V[id-1] - 2*d_V[id])
                                 / (dx*dx) +(d_V[id+(nx+2)] + d_V[id-(nx+2)]-2*d_V[id])/(dy*dy));

	}
}

void gpu(){
	int bpg;
	//tpb = 256;
	bpg = (nx+tpb-1)/tpb;
	boundary<<<bpg, tpb>>>(d_V);
        bpg = (nx*ny+tpb-1)/tpb;
	comp<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_it, d_d, d_f, d_cai, d_X, d_dV2);
	comp_dV2<<<bpg, tpb>>>(d_V, d_dV2, d_it);

	hipDeviceSynchronize();
}

__global__ void plane_waves(double *d_dV2){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<ny*5){
	int i, j;
	i = (int)(k/nx);
	j = k-i*nx;

	d_dV2[j*ny+i] = d_dV2[j*ny+i] + (-st);

	}
}

void stimu(){
	int bpg;
        //int tpb;

        //tpb = 256;
        bpg = (ny*5+tpb-1)/tpb;
	plane_waves<<<bpg, tpb>>>(d_dV2);
	hipDeviceSynchronize();
}

__global__ void Euler(double *d_V, double *d_dV2, double *d_Vnew, double *d_t){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){

	int j = (int)(k/nx);
	d_Vnew[k] = d_V[k+nx+2+1+2*j] + dt*d_dV2[k];
        d_V[k+nx+2+1+2*j] = d_Vnew[k];

	}

	if(k==0){

	d_t[0] = d_t[0] + dt;

	}
}

void Forward_Euler(){
	int bpg;
        //int tpb;

        //tpb = 256;
        bpg = (nx*ny+tpb-1)/tpb;
	Euler<<<bpg, tpb>>>(d_V, d_dV2, d_Vnew, d_t);
	hipDeviceSynchronize();
}
