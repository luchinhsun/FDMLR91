#include "head.h"

double *h_t;
double *d_t;
double *h_V;
double *d_V;
double *d_dV2;
double *h_Vnew;
double *d_Vnew;
double *d_it;

double *h_m;
double *d_m;
double *h_h;
double *d_h;
double *h_jj;
double *d_jj;
double *h_d;
double *d_d;
double *h_f;
double *d_f;
double *h_X;
double *d_X;
double *h_cai;
double *d_cai;

double *h_it;

double *d_m0;
double *d_h0;
double *d_jj0;
double *d_d0;
double *d_f0;
double *d_X0;

double *d_dVdt;
double *dcai;

void Allocate(){
	hipError_t Error;
	size_t size = nx*ny*sizeof(double);

	h_t = (double*)malloc(size);
	Error = hipMalloc((void**)&d_t, size);
	printf("CUDA error = %s\n",hipGetErrorString(Error));

	h_V = (double*)malloc((nx+2)*(ny+2)*sizeof(double));
	hipMalloc((void**)&d_V, (nx+2)*(ny+2)*sizeof(double));
	hipMalloc((void**)&d_dV2, size);
	h_Vnew = (double*)malloc(size);
	hipMalloc((void**)&d_Vnew, size);

	hipMalloc((void**)&d_it, size);

	h_m = (double*)malloc(size);
	hipMalloc((void**)&d_m, size);
	h_h = (double*)malloc(size);
        hipMalloc((void**)&d_h, size);
	h_jj = (double*)malloc(size);
        hipMalloc((void**)&d_jj, size);
	h_d = (double*)malloc(size);
        hipMalloc((void**)&d_d, size);
	h_f = (double*)malloc(size);
        hipMalloc((void**)&d_f, size);
	h_X = (double*)malloc(size);
        hipMalloc((void**)&d_X, size);
	h_cai = (double*)malloc(size);
        hipMalloc((void**)&d_cai, size);

	h_it = (double*)malloc(size);

	hipMalloc((void**)&d_m0, size);
        hipMalloc((void**)&d_h0, size);
        hipMalloc((void**)&d_jj0, size);
        hipMalloc((void**)&d_d0, size);
        hipMalloc((void**)&d_f0, size);
        hipMalloc((void**)&d_X0, size);

	hipMalloc((void**)&d_dVdt, size);
	hipMalloc((void**)&dcai, size);
}

void free(){

	free(h_t);free(h_V);free(h_m);free(h_h);
	free(h_jj);free(h_d);free(h_f);free(h_X);free(h_cai);
	free(h_Vnew);
	free(h_it);

	hipFree(d_t);hipFree(d_V);hipFree(d_dV2);hipFree(d_Vnew);hipFree(d_it);
	hipFree(d_m);hipFree(d_h);hipFree(d_jj);hipFree(d_d);
	hipFree(d_f);hipFree(d_X);hipFree(d_cai);

	hipFree(d_m0);hipFree(d_h0);hipFree(d_jj0);hipFree(d_d0);
        hipFree(d_f0);hipFree(d_X0);hipFree(d_dVdt);hipFree(dcai);
}

void Send_to_Device(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

	Error = hipMemcpy(d_t, h_t, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_t->d_t) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(d_V, h_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_V->d_V) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_m, h_m, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_m->d_m) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_h, h_h, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_h->d_h) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_jj, h_jj, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_jj->d_jj) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_d->d_d) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_f->d_f) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_X->d_X) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_cai, h_cai, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_cai->d_cai) = %s\n",hipGetErrorString(Error));
}

void Send_to_Host(){
	hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_V, d_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_m, d_m, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_m->h_m) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_h, d_h, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_h->h_h) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_jj, d_jj, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_jj->h_jj) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_d, d_d, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_d->h_d) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_f, d_f, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_f->h_f) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_X, d_X, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_X->h_X) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_cai, d_cai, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_cai->h_cai) = %s\n",hipGetErrorString(Error));
}

void Send_V(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);
/*
        Error = hipMemcpy(h_Vnew, d_Vnew, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Vnew->Vnew) = %s\n",hipGetErrorString(Error));
*/
	Error = hipMemcpy(h_Vnew, d_dVdt, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Vnew->Vnew) = %s\n",hipGetErrorString(Error));

}

void Save_Result(){

        FILE *pFile;
        int i,j;
        int index;
        //int n;
        //n = nx;
        pFile = fopen("V.txt","w+");
        // Save the matrix V
        for (i = 0; i < ny; i++) {
                for (j = 0; j < nx; j++) {
                        index = i*nx + j;
                        fprintf(pFile, "%g", h_Vnew[index]);
                        if (j == (nx-1)) {
                                fprintf(pFile, "\n");
                        }else{
                                fprintf(pFile, "\t");
                        }
                }
        }
        fclose(pFile);

	pFile = fopen("Vnotnew.txt","w+");
        // Save the matrix V
        for (i = 1; i < ny+1; i++) {
                for (j = 1; j < nx+1; j++) {
                        index = i*(nx+2) + j;
                        fprintf(pFile, "%g", h_V[index]);
                        if (j == (nx)) {
                                fprintf(pFile, "\n");
                        }else{
                                fprintf(pFile, "\t");
                        }
                }
        }
        fclose(pFile);
}

