#include "hip/hip_runtime.h"
#include "head.h"

#define tpb 256

extern float *d_t;
extern float *d_it;
extern float *d_V;
extern float *d_dV2;
extern float *d_Vnew;
extern float *d_m;
extern float *d_h;
extern float *d_jj;
extern float *d_d;
extern float *d_f;
extern float *d_X;
extern float *d_cai;

extern float *d_m0;
extern float *d_h0;
extern float *d_jj0;
extern float *d_d0;
extern float *d_f0;
extern float *d_X0;

extern float *d_dVdt;
extern float *dcai;

__global__ void boundary(float *d_V){
	int k = blockDim.x * blockIdx.x + threadIdx.x;

	if(k<nx){

	d_V[(k+1)*(nx+2)] = d_V[(k+1)*(nx+2)+1];
        d_V[(k+1)*(nx+2)+(nx+1)] = d_V[(k+1)*(nx+2)+nx];
        d_V[k+1] = d_V[k+1+(nx+2)];
        d_V[(ny+1)*(nx+2)+k+1] = d_V[ny*(nx+2)+k+1];

	}
}

void bc(){
        int bpg;
        //tpb = 256;
        bpg = (nx+tpb-1)/tpb;
        boundary<<<bpg, tpb>>>(d_V);
        //hipDeviceSynchronize();
}


__global__ void comp_dV2(float *d_V ,float *d_dV2){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){

	int i = (int)(k/nx);
	int id = k+(nx+2)+1+(2*i);

	d_dV2[k] = D*((d_V[id+1] + d_V[id-1] - 2*d_V[id]) / (dx*dx) + (d_V[id+(nx+2)] + d_V[id-(nx+2)] - 2*d_V[id])/(dy*dy));

	}
}

void dV2(){
	int bpg;
	//tpb = 256;
        bpg = (nx*ny+tpb-1)/tpb;
	comp_dV2<<<bpg, tpb>>>(d_V, d_dV2);
	//hipDeviceSynchronize();
}

__device__ void comp_it(float *d_V, float *d_m, float *d_h, float *d_jj, float *d_d, float *d_f, float *d_cai, float *dcai, float *d_X, float *d_it, float *d_m0, float *d_h0, float *d_jj0, float *d_d0, float *d_f0, float *d_X0, int I, int i, int k, float *d_t) {
	//int id = k+nx+2+1+2*j;
	d_it[k] = 0.0;

	//comp_ina
	float gna = 23;
        float ena = ((R*temp) / frdy)*__logf(nao / nai);

        float am = 0.32*(d_V[k+nx+2+1+2*i] + 47.13) / (1 - __expf(-0.1*(d_V[k+nx+2+1+2*i] + 47.13)));
        float bm = 0.08*__expf(-d_V[k+nx+2+1+2*i] / 11);
	float ah, bh, aj ,bj;
        if (d_V[k+nx+2+1+2*i] < -40.0) {
                ah = 0.135*__expf((80 + d_V[k+nx+2+1+2*i]) / -6.8);
                bh = 3.56*__expf(0.079*d_V[k+nx+2+1+2*i]) + 310000 * __expf(0.35*d_V[k+nx+2+1+2*i]);
                aj = (-127140 * __expf(0.2444*d_V[k+nx+2+1+2*i]) - 0.00003474*__expf(-0.04391*d_V[k+nx+2+1+2*i]))*
                        ((d_V[k+nx+2+1+2*i] + 37.78)/(1 + __expf(0.311*(d_V[k+nx+2+1+2*i] + 79.23))));
                bj = (0.1212*__expf(-0.01052*d_V[k+nx+2+1+2*i])) / (1 + __expf(-0.1378*(d_V[k+nx+2+1+2*i] + 40.14)));
        }
        else {
                ah = 0;
                bh = 1 / (0.13*(1 + __expf((d_V[k+nx+2+1+2*i] + 10.66) / -11.1)));
                aj = 0;
                bj = (0.3*__expf(-0.0000002535*d_V[k+nx+2+1+2*i])) / (1 + __expf(-0.1*(d_V[k+nx+2+1+2*i] + 32)));
        }
        float mtau = 1 / (am + bm);
        float htau = 1 / (ah + bh);
	float jtau = 1 / (aj + bj);

        float mss = am*mtau;
        float hss = ah*htau;
        float jss = aj*jtau;

        d_m0[k] = mss - (mss - d_m[k])*__expf(-d_t[k] / mtau);
        d_h0[k] = hss - (hss - d_h[k])*__expf(-d_t[k] / htau);
        d_jj0[k] = jss - (jss - d_jj[k])*__expf(-d_t[k] / jtau);

        d_it[k] += gna*d_m0[k] * d_m0[k] * d_m0[k] * d_h0[k] * d_jj0[k] * (d_V[k+nx+2+1+2*i] - ena);
	//comp_ical
	__shared__ float esi[tpb];
	__shared__ float isi[tpb];
        esi[I] = 7.7 - 13.0287*__logf(d_cai[k]);

        float ad = 50 * 0.095*__expf(-0.01*(d_V[k+nx+2+1+2*i] - 5)) / (1 + __expf(-0.072*(d_V[k+nx+2+1+2*i] - 5)));
        float bd = 50 * 0.07*__expf(-0.017*(d_V[k+nx+2+1+2*i] + 44)) / (1 + __expf(0.05*(d_V[k+nx+2+1+2*i] + 44)));
        float af = 50 * 0.012*__expf(-0.008*(d_V[k+nx+2+1+2*i] + 28)) / (1 + __expf(0.15*(d_V[k+nx+2+1+2*i] + 28)));
        float bf = 50 * 0.0065*__expf(-0.02*(d_V[k+nx+2+1+2*i] + 30)) / (1 + __expf(-0.2*(d_V[k+nx+2+1+2*i] + 30)));

        float taud = 1 / (ad + bd);
        float tauf = 1 / (af + bf);

        float dss = ad*taud;
        float fss = af*tauf;

        d_d0[k] = dss - (dss - d_d[k])*__expf(-d_t[k] / taud);
        d_f0[k] = fss - (fss - d_f[k])*__expf(-d_t[k] / tauf);

        isi[I] = 0.09*d_d0[k] * d_f0[k] * (d_V[k+nx+2+1+2*i] - esi[I]);

        dcai[k] = -0.0001*isi[I] + 0.07*(0.0001 - d_cai[k]);

        //d_cai[k] = d_cai[k] + dcai*dt;
	d_it[k] = d_it[k] + isi[I];
	//comp_ik
        float gk = 0.282*sqrt(ko / 5.4);
        float ek = ((R*temp) / frdy)*__logf(ko / ki);
        //float prnak = 0.01833;
        //ek = ((R*temp) / frdy)*__logf((ko + prnak*nao) / (ki + prnak*nai));

        float ax = 50 * 0.0005*__expf(0.083*(d_V[k+nx+2+1+2*i] + 50)) / (1 + __expf(0.057*(d_V[k+nx+2+1+2*i] + 50)));
        float bx = 50 * 0.0013*__expf(-0.06*(d_V[k+nx+2+1+2*i] + 20)) / (1 + __expf(-0.04*(d_V[k+nx+2+1+2*i] + 20)));

        float taux = 1 / (ax + bx);
        float xss = ax*taux;
        d_X0[k] = xss - (xss - d_X[k])*__expf(-d_t[k] / taux);

	float Xi;
        if (d_V[k+nx+2+1+2*i] > -100) {
                Xi = 2.837*(__expf(0.04*(d_V[k+nx+2+1+2*i] + 77)) - 1)/((d_V[k+nx+2+1+2*i] + 77 + 1e-15)*__expf(0.04*(d_V[k+nx+2+1+2*i] + 35)));
        }
        else {
                Xi = 1;
        }
        d_it[k] += gk*d_X0[k] * Xi*(d_V[k+nx+2+1+2*i] - ek);
	//comp_ik1
        float gk1 = 0.6047*(sqrt(ko / 5.4));
        float ek1 = ((R*temp) / frdy)*__logf(ko / ki);

        float ak1 = 1.02 / (1 + __expf(0.2385*(d_V[k+nx+2+1+2*i] - ek1 - 59.215)));
        float bk1 = (0.49124*__expf(0.08032*(d_V[k+nx+2+1+2*i] - ek1 + 5.476))+__expf(0.06175*(d_V[k+nx+2+1+2*i] - ek1 - 594.31)))/(1 + __expf(-0.5143*(d_V[k+nx+2+1+2*i] - ek1 + 4.753)));
        float K1ss = ak1 / (ak1 + bk1);

        d_it[k] += gk1*K1ss*(d_V[k+nx+2+1+2*i] - ek1);
	//comp_ikp
        float gkp = 0.0183;
        float ekp = ((R*temp) / frdy)*__logf(ko / ki);

        float kp = 1 / (1 + __expf((7.488 - d_V[k+nx+2+1+2*i]) / 5.98));

        d_it[k] += gkp*kp*(d_V[k+nx+2+1+2*i] - ekp);
	//comp_ib
        d_it[k] += 0.03921*(d_V[k+nx+2+1+2*i] + 59.87);

}

__global__ void comp_dVdt(float *d_V, float *d_m, float *d_h, float *d_jj, float *d_d, float *d_f, float *d_cai, float *dcai, float *d_X, float *d_it, float *d_m0, float *d_h0, float *d_jj0, float *d_d0, float *d_f0, float *d_X0, float *d_dVdt, float *d_t){

	int k = threadIdx.x + blockIdx.x * blockDim.x;
        int I = threadIdx.x;

        if(k<nx*ny){

        int i = (int)(k/nx);
	d_t[k] = dt_max;
	comp_it(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, I, i, k, d_t);
	d_dVdt[k] = -d_it[k];
	}
}

void dVdt(){
	int bpg;

        bpg = (nx*ny+tpb-1)/tpb;
        comp_dVdt<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, d_dVdt, d_t);
}

__global__ void plane_waves(float *d_dVdt){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<ny*5){
	int i, j, id;
	i = (int)(k/5);
	j = k-i*5;
	id = i*nx+j;

	d_dVdt[id] = d_dVdt[id] + (-st);

	}
}

void stimu(){
	int bpg;
        //int tpb;

        //tpb = 256;
        bpg = (ny*5+tpb-1)/tpb;
	plane_waves<<<bpg, tpb>>>(d_dVdt);
	//hipDeviceSynchronize();
}


__device__ void gate(float *d_m, float *d_h, float *d_jj, float *d_d, float *d_f, float *d_X, float *d_m0, float *d_h0, float *d_jj0, float *d_d0, float *d_f0, float *d_X0, int k){
	d_m[k] = d_m0[k];
	d_h[k] = d_h0[k];
	d_jj[k] = d_jj0[k];
	d_d[k] = d_d0[k];
	d_f[k] = d_f0[k];
	d_X[k] = d_X0[k];
}

__global__ void comp_ODE_stim(float *d_V, float *d_m, float *d_h, float *d_jj, float *d_d, float *d_f, float *d_cai, float *dcai, float *d_X, float *d_it, float *d_m0, float *d_h0, float *d_jj0, float *d_d0, float *d_f0, float *d_X0, float *d_dVdt, float *d_t){

	int k = threadIdx.x + blockIdx.x * blockDim.x;
        int I = threadIdx.x;

        if(k<nx*ny){

        int i = (int)(k/nx);
	int j = k - i*nx;
	int id = i*nx+j;
	int k1, k0, ttt;
	int vid = (i+1)*(nx+2)+j+1;

	if(d_dVdt[id]>0){
		k0 = 5;
	}else{
		k0 = 1;
	}
	k1 = k0 + (int)(fabs(d_dVdt[id]) + 0.5);
	if (k1 >(int)(dt_max / dt_min)){
		k1 = (int)(dt_max / dt_min);
	}
	d_t[id] = dt_max / k1;
	for (ttt = 0; ttt < k1; ttt++){ //from t to t+dt_max, t=t+dt
		comp_it(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, I, i, id, d_t);
		gate(d_m, d_h, d_jj, d_d, d_f, d_X, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, id);
		d_cai[id] = d_cai[id] + dcai[id]*d_t[id];//renew Cai
		if(i>0 && i<5){
			d_dVdt[id] = -d_it[id] + (-st);
		}else{
			d_dVdt[id] = -d_it[id];
		}
		d_V[vid] = d_V[vid] + d_t[id]*d_dVdt[id];
        }

	}
}

__global__ void comp_ODE(float *d_V, float *d_m, float *d_h, float *d_jj, float *d_d, float *d_f, float *d_cai, float *dcai, float *d_X, float *d_it, float *d_m0, float *d_h0, float *d_jj0, float *d_d0, float *d_f0, float *d_X0, float *d_dVdt, float *d_t){

        int k = threadIdx.x + blockIdx.x * blockDim.x;
        int I = threadIdx.x;

        if(k<nx*ny){

        int i = (int)(k/nx);
        int j = k - i*nx;
        int id = i*nx+j;
        int k1, k0, ttt;
	int vid = (i+1)*(nx+2)+j+1;

        if(d_dVdt[id]>0){
                k0 = 5;
        }else{
                k0 = 1;
        }
        k1 = k0 + (int)(fabs(d_dVdt[id])+0.5);
        if (k1 >(int)(dt_max / dt_min)){
                k1 = (int)(dt_max / dt_min);
        }
        d_t[id] = dt_max / k1;
        for (ttt = 0; ttt < k1; ttt++){ //from t to t+dt_max, t=t+dt
                comp_it(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, I, i, id, d_t);
                gate(d_m, d_h, d_jj, d_d, d_f, d_X, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, id);
                d_cai[id] = d_cai[id] + dcai[id]*d_t[id];//renew Cai
                d_dVdt[id] = -d_it[id];
                d_V[vid] = d_V[vid] + d_t[id]*d_dVdt[id];
        }

        }
}

void ODE_stim(){
	int bpg;
        bpg = (nx*ny+tpb-1)/tpb;
        comp_ODE_stim<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, d_dVdt, d_t);
//	bpg = ((nx-5)*ny+tpb-1)/tpb;
//	comp_ODE<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, d_dVdt, d_t, 5);
}

void ODE(){
	int bpg;
        bpg = (nx*ny+tpb-1)/tpb;
        comp_ODE<<<bpg, tpb>>>(d_V, d_m, d_h, d_jj, d_d, d_f, d_cai, dcai, d_X, d_it, d_m0, d_h0, d_jj0, d_d0, d_f0, d_X0, d_dVdt, d_t);
}

__global__ void Euler(float *d_V, float *d_dV2, float *d_Vnew){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){

	int i = (int)(k/nx);
	d_Vnew[k] = d_V[k+nx+2+1+2*i] + dt_max/2 *d_dV2[k];
        d_V[k+nx+2+1+2*i] = d_Vnew[k];

	}
}

void Forward_Euler(){
	int bpg;
        //int tpb;

        //tpb = 256;
        bpg = (nx*ny+tpb-1)/tpb;
	Euler<<<bpg, tpb>>>(d_V, d_dV2, d_Vnew);
	//hipDeviceSynchronize();
}
