#include "head.h"

float *h_t;
float *d_t;
float *h_V;
float *d_V;
float *d_dV2;
float *h_Vnew;
float *d_Vnew;
float *d_it;

float *h_m;
float *d_m;
float *h_h;
float *d_h;
float *h_jj;
float *d_jj;
float *h_d;
float *d_d;
float *h_f;
float *d_f;
float *h_X;
float *d_X;
float *h_cai;
float *d_cai;

float *h_it;

float *d_m0;
float *d_h0;
float *d_jj0;
float *d_d0;
float *d_f0;
float *d_X0;

float *d_dVdt;
float *dcai;

void Allocate(){
	hipError_t Error;
	size_t size = nx*ny*sizeof(float);

	h_t = (float*)malloc(size);
	Error = hipMalloc((void**)&d_t, size);
	printf("CUDA error = %s\n",hipGetErrorString(Error));

	h_V = (float*)malloc((nx+2)*(ny+2)*sizeof(float));
	hipMalloc((void**)&d_V, (nx+2)*(ny+2)*sizeof(float));
	hipMalloc((void**)&d_dV2, size);
	h_Vnew = (float*)malloc(size);
	hipMalloc((void**)&d_Vnew, size);

	hipMalloc((void**)&d_it, size);

	h_m = (float*)malloc(size);
	hipMalloc((void**)&d_m, size);
	h_h = (float*)malloc(size);
        hipMalloc((void**)&d_h, size);
	h_jj = (float*)malloc(size);
        hipMalloc((void**)&d_jj, size);
	h_d = (float*)malloc(size);
        hipMalloc((void**)&d_d, size);
	h_f = (float*)malloc(size);
        hipMalloc((void**)&d_f, size);
	h_X = (float*)malloc(size);
        hipMalloc((void**)&d_X, size);
	h_cai = (float*)malloc(size);
        hipMalloc((void**)&d_cai, size);

	h_it = (float*)malloc(size);

	hipMalloc((void**)&d_m0, size);
        hipMalloc((void**)&d_h0, size);
        hipMalloc((void**)&d_jj0, size);
        hipMalloc((void**)&d_d0, size);
        hipMalloc((void**)&d_f0, size);
        hipMalloc((void**)&d_X0, size);

	hipMalloc((void**)&d_dVdt, size);
	hipMalloc((void**)&dcai, size);
}

void free(){

	free(h_t);free(h_V);free(h_m);free(h_h);
	free(h_jj);free(h_d);free(h_f);free(h_X);free(h_cai);
	free(h_Vnew);
	free(h_it);

	hipFree(d_t);hipFree(d_V);hipFree(d_dV2);hipFree(d_Vnew);hipFree(d_it);
	hipFree(d_m);hipFree(d_h);hipFree(d_jj);hipFree(d_d);
	hipFree(d_f);hipFree(d_X);hipFree(d_cai);

	hipFree(d_m0);hipFree(d_h0);hipFree(d_jj0);hipFree(d_d0);
        hipFree(d_f0);hipFree(d_X0);hipFree(d_dVdt);hipFree(dcai);
}

void Send_to_Device(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(float);

	Error = hipMemcpy(d_t, h_t, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_t->d_t) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(d_V, h_V, (nx+2)*(ny+2)*sizeof(float), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_V->d_V) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_m, h_m, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_m->d_m) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_h, h_h, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_h->d_h) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_jj, h_jj, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_jj->d_jj) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_d->d_d) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_f->d_f) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_X->d_X) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_cai, h_cai, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_cai->d_cai) = %s\n",hipGetErrorString(Error));
}

void Send_to_Host(){
	hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(float);

        Error = hipMemcpy(h_V, d_V, (nx+2)*(ny+2)*sizeof(float), hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_m, d_m, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_m->h_m) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_h, d_h, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_h->h_h) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_jj, d_jj, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_jj->h_jj) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_d, d_d, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_d->h_d) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_f, d_f, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_f->h_f) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_X, d_X, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_X->h_X) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_cai, d_cai, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_cai->h_cai) = %s\n",hipGetErrorString(Error));
}

void Send_V(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(float);

        Error = hipMemcpy(h_Vnew, d_Vnew, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Vnew->Vnew) = %s\n",hipGetErrorString(Error));

}

void Save_Result(){

        FILE *pFile;
        int i,j;
        int index;
        //int n;
        //n = nx;
        pFile = fopen("V.txt","w+");
        // Save the matrix V
        for (i = 0; i < ny; i++) {
                for (j = 0; j < nx; j++) {
                        index = i*nx + j;
                        fprintf(pFile, "%g", h_Vnew[index]);
                        if (j == (nx-1)) {
                                fprintf(pFile, "\n");
                        }else{
                                fprintf(pFile, "\t");
                        }
                }
        }
        fclose(pFile);
/*
	pFile = fopen("Vnotnew.txt","w+");
        // Save the matrix V
        for (i = 1; i < ny+1; i++) {
                for (j = 1; j < nx+1; j++) {
                        index = i*(nx+2) + j;
                        fprintf(pFile, "%g", h_V[index]);
                        if (j == (nx)) {
                                fprintf(pFile, "\n");
                        }else{
                                fprintf(pFile, "\t");
                        }
                }
        }
        fclose(pFile);*/
}

